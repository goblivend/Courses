#include "hip/hip_runtime.h"
#include "Compute.hpp"
#include "Image.hpp"
#include "logo.h"

// Single threaded version of the Method
__global__ void mykernel(ImageView<rgb8> in, ImageView<uint8_t> logo)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < in.width && y < in.height)
    {
        rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
        pixel[x].r = 0;

        if (x < logo.width && y < logo.height)
        {
            float alpha = logo.buffer[y * logo.stride + x] / 255.f;
            pixel[x].g = uint8_t(alpha * pixel[x].g + (1 - alpha) * 255);
            pixel[x].b = uint8_t(alpha * pixel[x].b + (1 - alpha) * 255);
        }
    }
}


void compute_cu(ImageView<rgb8> in)
{
    static Image<uint8_t> device_logo;

    dim3 block(16, 16);
    dim3 grid((in.width + block.x - 1) / block.x, (in.height + block.y - 1) / block.y);
    
    // Copy the logo to the device if it is not already there
    if (device_logo.buffer == nullptr)
    {
        device_logo = Image<uint8_t>(logo_width, logo_height, true);
        hipMemcpy2D(device_logo.buffer, device_logo.stride, logo_data, logo_width, logo_width, logo_height, hipMemcpyHostToDevice);
    }

    // Copy the input image to the device
    Image<rgb8> device_in(in.width, in.height, true);
    hipMemcpy2D(device_in.buffer, device_in.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);
    
    mykernel<<<grid, block>>>(device_in, device_logo);

    // Copy the result back to the host
    hipMemcpy2D(in.buffer, in.stride, device_in.buffer, device_in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToHost);
}