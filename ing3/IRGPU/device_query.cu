
#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char **argv) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  std::cerr << "Getting GPU Data." << std::endl;
  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        std::cerr << "No CUDA GPU has been detected" << std::endl;
        return -1;
      } else if (deviceCount == 1) {
        std::cerr << "There is 1 device supporting CUDA" << std::endl;
      } else {
        std::cerr << "There are " << deviceCount << " devices supporting CUDA"
                  << std::endl;
      }
    }
    std::cerr << "Device " << dev
              << " name: " << deviceProp.name //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Computational Capabilities: "
              << deviceProp.major //@@ Print appropriate `deviceProp` field
              << "."
              << deviceProp.minor //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum global memory size: "
              << deviceProp.totalGlobalMem //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum constant memory size: "
              << deviceProp.totalConstMem //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum shared memory size per block: "
              << deviceProp.sharedMemPerBlock //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum block dimensions: "
              << deviceProp.maxThreadsDim[0] //@@ Print appropriate `deviceProp` field
              << " x " << deviceProp.maxThreadsDim[1] //@@ Print appropriate `deviceProp` field
              << " x " << deviceProp.maxThreadsDim[2] //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Maximum grid dimensions: "
              << deviceProp.maxGridSize[0] //@@ Print appropriate `deviceProp` field
              << " x " << deviceProp.maxGridSize[1] //@@ Print appropriate `deviceProp` field
              << " x " << deviceProp.maxGridSize[2] //@@ Print appropriate `deviceProp` field
              << std::endl;
    std::cerr << " Warp size: "
              << deviceProp.warpSize //@@ Print appropriate `deviceProp` field
              << std::endl;
  }
  std::cerr << "End of GPU data gathering." << std::endl;
  return 0;
}
