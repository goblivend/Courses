#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>

[[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)


struct rgba8_t {
  std::uint8_t r;
  std::uint8_t g;
  std::uint8_t b;
  std::uint8_t a;
};

__device__ rgba8_t heat_lut(float x)
{
  assert(0 <= x && x <= 1);
  float x0 = 1.f / 4.f;
  float x1 = 2.f / 4.f;
  float x2 = 3.f / 4.f;

  if (x < x0)
  {
    auto g = static_cast<std::uint8_t>(x / x0 * 255);
    return rgba8_t{0, g, 255, 255};
  }
  else if (x < x1)
  {
    auto b = static_cast<std::uint8_t>((x1 - x) / x0 * 255);
    return rgba8_t{0, 255, b, 255};
  }
  else if (x < x2)
  {
    auto r = static_cast<std::uint8_t>((x - x1) / x0 * 255);
    return rgba8_t{r, 255, 0, 255};
  }
  else
  {
    auto b = static_cast<std::uint8_t>((1.f - x) / x0 * 255);
    return rgba8_t{255, b, 0, 255};
  }
}

__device__ float2 im_square(float2 z)
{
  return {z.x * z.x - z.y * z.y, 2 * z.x * z.y};
}


__device__ rgba8_t spectral_color(double l) // RGB <0,1> <- lambda l <400,700> [nm]
    {
      if (l < 0.25) {
        return {0, static_cast<std::uint8_t>(l / 0.25 * 255), 255, 255};
      } else if (l < 0.5) {
        return {0, 255, static_cast<std::uint8_t>((0.5 - l) / 0.25 * 255), 255};
      } else if (l < 0.75) {
        return {static_cast<std::uint8_t>((l - 0.5) / 0.25 * 255), 255, 0, 255};
      } else if (l < 1) {
        return {255, static_cast<std::uint8_t>((1 - l) / 0.25 * 255), 0, 255};
      } else {
        return {0, 0, 0, 255};
      }
    }


// Device code
__global__ void mykernel(char* buffer, int width, int height, size_t pitch, int n_iterations)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  uchar4*  lineptr = (uchar4*)(buffer + y * pitch);


  float2 centerPoint = {-0.75f, 0.f};
  float2 size = {1.75f, 1.f};

  float2 centered = {x - width / 2.f, y - height / 2.f};
  float2 unit = {centered.x / width * 2.f, centered.y / height * 2.f};
  float2 z = {unit.x * size.x + centerPoint.x, unit.y * size.y + centerPoint.y};

  float2 z2 = {0.f, 0.f};

  int i = 0;
  while (i < n_iterations) {
    z2 = im_square(z2);
    z2.x += z.x;
    z2.y += z.y;
    if (z2.x * z2.x + z2.y * z2.y > 4)
      break;
    i++;
  }

  float t = (float)i / n_iterations;


  // rgba8_t color = heat_lut(t);
  rgba8_t color = spectral_color(t);

  lineptr[x]    = {color.r, color.g, color.b, color.a};
  }









/// Compute the number or iteration of the fractal per pixel and store the result in *buffer*.
/// Note that a 32-bits location can be used to store an integer (int32) or a color (uchar4).
///
/// \param buffer Input buffer of type (uchar4 or uint32_t)
/// \param width Width of the image
/// \param height Height of the image
/// \param pitch Size of a line in bytes
/// \param max_iter Maximum number of iterations
__global__ void compute_iter(char* buffer, int width, int height, size_t pitch, int max_iter) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  uint32_t*  lineptr = (uint32_t*)(buffer + y * pitch);


  float2 centerPoint = {-0.75f, 0.f};
  float2 size = {1.75f, 1.f};

  float2 centered = {x - width / 2.f, y - height / 2.f};
  float2 unit = {centered.x / width * 2.f, centered.y / height * 2.f};
  float2 z = {unit.x * size.x + centerPoint.x, unit.y * size.y + centerPoint.y};

  float2 z2 = {0.f, 0.f};

  int i = 0;
  while (i < max_iter) {
    z2 = im_square(z2);
    z2.x += z.x;
    z2.y += z.y;
    if (z2.x * z2.x + z2.y * z2.y > 4)
      break;
    i++;
  }

  lineptr[x] = i;
}


/// compute histo
///
/// \param buffer Input buffer of type (uchar4 or uint32_t)
/// \param width Width of the image
/// \param height Height of the image
/// \param pitch Size of a line in bytes
/// \param max_iter Maximum number of iterations
/// \param LUT Output look-up table
__global__ void compute_histo(const char* buffer, int width, int height, size_t pitch, int max_iter, uchar4* LUT) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  if (x >= width || y >= height)
    return;


  // Compute the histogram
  uint32_t *histo = (uint32_t *) LUT;
  uint32_t *lineptr = (uint32_t*)(buffer + y * pitch);


  atomicAdd(histo + lineptr[x], 1);
}


/// This function is single thread for now!
///
/// \param width Width of the image
/// \param height Height of the image
/// \param pitch Size of a line in bytes
/// \param max_iter Maximum number of iterations
/// \param LUT Output look-up table
__global__ void compute_LUT(int width, int height, size_t pitch, int max_iter, uchar4* LUT) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  if (x > 0)
    return;

  uint32_t *histo = (uint32_t *) LUT;


  // Compute the cumulative distribution function
  for (int i = 1; i < max_iter; i++) {
    histo[i] = histo[i - 1] + histo[i];
  }

  // rgba8_t *lut_ptr = (uchar4*)LUT;
  for (int i = 0; i < max_iter; i++) {
    rgba8_t color = heat_lut((float)histo[i] / histo[max_iter - 1]);
    LUT[i] = {color.r, color.g, color.b, 255};
  }

  LUT[max_iter] = {0, 0, 0, 255};
}


///
/// \param buffer Input buffer of type (uchar4 or uint32_t)
/// \param width Width of the image
/// \param height Height of the image
/// \param pitch Size of a line in bytes
/// \param max_iter Maximum number of iterations
__global__ void apply_LUT(char* buffer, int width, int height, size_t pitch, int max_iter, const uchar4* LUT) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  uint32_t*  lineptr = (uint32_t*)(buffer + y * pitch);
  uint32_t *lut_ptr = (uint32_t*)LUT;
  lineptr[x] = lut_ptr[lineptr[x]];
}


void render(char* hostBuffer, int width, int height, std::ptrdiff_t stride, int n_iterations)
{
  hipError_t rc = hipSuccess;

  // Allocate device memory
  char*  devBuffer;
  size_t pitch;

  rc = hipMallocPitch(&devBuffer, &pitch, width * sizeof(rgba8_t), height);
  if (rc)
    abortError("Fail buffer allocation");

  // Run the kernel with blocks of size 64 x 64
  {
    int bsize = 32;
    int w     = std::ceil((float)width / bsize);
    int h     = std::ceil((float)height / bsize);

    spdlog::debug("running kernel of size ({},{})", w, h);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);
    // mykernel<<<dimGrid, dimBlock>>>(devBuffer, width, height, pitch, n_iterations);


    uchar4* lut;
    size_t pitch_lut;
    rc = hipMallocPitch(&lut, &pitch_lut, (n_iterations + 1) * sizeof(uchar4), 1);
    if (rc)
      abortError("Fail buffer allocation");


    compute_iter<<<dimGrid, dimBlock>>>(devBuffer, width, height, pitch, n_iterations);
    if (hipPeekAtLastError())
      abortError("Computation Error");

    compute_histo<<<dimGrid, dimBlock>>>(devBuffer, width, height, pitch, n_iterations, lut);
    if (hipPeekAtLastError())
      abortError("Computation Error");

    compute_LUT<<<1, 1>>>(width, height, pitch, n_iterations, lut);
    if (hipPeekAtLastError())
      abortError("Computation Error");

    apply_LUT<<<dimGrid, dimBlock>>>(devBuffer, width, height, pitch, n_iterations, lut);
    if (hipPeekAtLastError())
      abortError("Computation Error");
  }

  // Copy back to main memory
  rc = hipMemcpy2D(hostBuffer, stride, devBuffer, pitch, width * sizeof(rgba8_t), height, hipMemcpyDeviceToHost);
  if (rc)
    abortError("Unable to copy buffer back to memory");

  // Free
  rc = hipFree(devBuffer);
  if (rc)
    abortError("Unable to free memory");
}
